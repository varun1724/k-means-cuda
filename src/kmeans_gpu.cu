#include "hip/hip_runtime.h"
#include "../include/kmeans_gpu.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define CUDA_CHECK(x) do { \
    hipError_t e = x; \
    if (e != hipSuccess) { \
        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

// Initialize centroids on GPU
__global__ void initializeCentroidsGPU(float* points, float* centroids, int num_points, int num_centroids, int dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_centroids) {
        int point_idx = (idx * num_points) / num_centroids;
        for (int d = 0; d < dim; d++) {
            centroids[idx * dim + d] = points[point_idx * dim + d];
        }
    }
}

// Assign points to clusters on GPU
__global__ void assignPointsGPU(float* points, float* centroids, int* clusters, 
                               int num_points, int num_centroids, int dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_points) {
        float min_dist = INFINITY;
        int closest_centroid = 0;
        
        // Find the nearest centroid for the current point
        for (int c = 0; c < num_centroids; c++) {
            float dist = 0.0f;
            for (int d = 0; d < dim; d++) {
                float diff = points[idx * dim + d] - centroids[c * dim + d];
                dist += diff * diff;
            }
            
            if (dist < min_dist) {
                min_dist = dist;
                closest_centroid = c;
            }
        }
        
        clusters[idx] = closest_centroid;
    }
}

// Assign points to clusters for a specific dimension on GPU
__global__ void updatePointsForDimGPU(float* points, int* clusters, int* counts, float* sums, int num_points, int num_dims, int curDim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_points) {
        int cluster = clusters[idx];
        atomicAdd(&sums[cluster * num_dims + curDim], points[idx * num_dims + curDim]);
        if (curDim == 0) {
            atomicAdd(&counts[cluster], 1);
        }
    }
}

// Update centroids on GPU - accumulate sums and counts
__global__ void updateCentroidsGPU(float* points, float* centroids, float* newCentroids, 
                                  int* clusters, int* counts, float* sums, int num_points, int num_centroids, int dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < num_centroids) {
        for (int d = 0; d < dim; d++) {
            if (counts[idx] > 0) {
                newCentroids[idx * dim + d] = sums[idx * dim + d] / counts[idx];
            } else {
                newCentroids[idx * dim + d] = centroids[idx * dim + d];
            }
        }
    }
}

// Main k-means function
bool kmeans_cuda(float* points, float* centroids, int* clusters,
                int num_points, int num_centroids, int dim, int max_iterations, float tolerance,
                int* iterations) {
    float *d_points, *d_centroids, *d_newCentroids, *d_sums;
    int *d_clusters, *d_counts;
    bool converged = false;
    *iterations = 0;
    
    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_points, num_points * dim * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_centroids, num_centroids * dim * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_newCentroids, (num_centroids * dim) * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_clusters, num_points * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_counts, num_centroids * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_sums, num_centroids * dim * sizeof(float)));
    
    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_points, points, num_points * dim * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_centroids, centroids, num_centroids * dim * sizeof(float), hipMemcpyHostToDevice));
    
    // Calculate grid and block dimensions
    int block_size = 256;
    int num_blocks_points = (num_points + block_size - 1) / block_size;
    int num_blocks_centroids = (num_centroids + block_size - 1) / block_size;
    
    // Initialize centroids
    initializeCentroidsGPU<<<num_blocks_centroids, block_size>>>(d_points, d_centroids, num_points, num_centroids, dim);
    CUDA_CHECK(hipDeviceSynchronize());
    
    do {
        (*iterations)++;
        
        // Reset arrays
        CUDA_CHECK(hipMemset(d_newCentroids, 0, (num_centroids * dim) * sizeof(float)));
        CUDA_CHECK(hipMemset(d_counts, 0, num_centroids * sizeof(int)));
        CUDA_CHECK(hipMemset(d_sums, 0, num_centroids * dim * sizeof(float)));
        
        // Assign points to nearest centroids
        assignPointsGPU<<<num_blocks_points, block_size>>>(d_points, d_centroids, d_clusters, num_points, num_centroids, dim);
        CUDA_CHECK(hipDeviceSynchronize());

        // Update points to clusters for each dimension
        for (int d = 0; d < dim; d++) {
            updatePointsForDimGPU<<<num_blocks_points, block_size>>>(d_points, d_clusters, d_counts, d_sums, num_points, dim, d);
            CUDA_CHECK(hipDeviceSynchronize());
        }

        // Update centroids
        updateCentroidsGPU<<<num_blocks_points, block_size>>>(d_points, d_centroids, d_newCentroids, d_clusters, d_counts, d_sums, num_points, num_centroids, dim);
        CUDA_CHECK(hipDeviceSynchronize());
        
        // Copy new centroids back to host for convergence check
        float* newCentroids = new float[num_centroids * dim];
        CUDA_CHECK(hipMemcpy(newCentroids, d_newCentroids, (num_centroids * dim) * sizeof(float), hipMemcpyDeviceToHost));
        
        // Check convergence
        float max_diff = 0.0f;
        for (int c = 0; c < num_centroids; c++) {
            for (int d = 0; d < dim; d++) {
                int idx = c * dim + d;
                float diff = fabs(centroids[idx] - newCentroids[idx]);
                if (diff > max_diff) {
                    max_diff = diff;
                }
            }
        }

        printf("Max diff: %.6f, Iteration: %d, Tolerance: %.3f\n", max_diff, *iterations, tolerance);

        // Update centroids for next iteration
        for (int i = 0; i < num_centroids * dim; i++) {
            centroids[i] = newCentroids[i];
        }
        
        // Copy updated centroids back to device
        CUDA_CHECK(hipMemcpy(d_centroids, centroids, num_centroids * dim * sizeof(float), hipMemcpyHostToDevice));
        
        delete[] newCentroids;
        
        if (max_diff < tolerance) {
            converged = true;
            break;
        }
        
    } while (*iterations < max_iterations);
    
    // Copy final results back to host
    CUDA_CHECK(hipMemcpy(clusters, d_clusters, num_points * sizeof(int), hipMemcpyDeviceToHost));
    
    // Free device memory
    CUDA_CHECK(hipFree(d_points));
    CUDA_CHECK(hipFree(d_centroids));
    CUDA_CHECK(hipFree(d_newCentroids));
    CUDA_CHECK(hipFree(d_clusters));
    CUDA_CHECK(hipFree(d_counts));
    CUDA_CHECK(hipFree(d_sums));
    return converged;
} 